﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <time.h>

__global__ void addKernel(int* c, const int* a, const int* b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void multiplyKernel(int* c, const int* a, const int* b) {
    int i = threadIdx.x;
    c[i] = a[i] * b[i];
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t mathWithCuda(int* c, const int* a, const int* b, unsigned int size, void (*operation)(int*, const int*, const int*)) {
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    operation <<< 1, size >>> (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

int cudaTearDown(hipError_t cudaStatus) {

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

}

// Helper function to add vectors in series (CPU)
int addVectors(const int a[], const int b[], int c[], int arraySize) {
    for (int i = 0; i < arraySize; i++) {
        c[i] = a[i] + b[i];
    }
    return 0;
}

// Helper function to multiply vectors in series (CPU)
int multiplyVectors(const int a[], const int b[], int c[], int arraySize) {
    for (int i = 0; i < arraySize; i++) {
        c[i] = a[i] * b[i];
    }
    return 0;
}

static void printArray(int arr[], int arraySize) {
    for (int i = 0; i < arraySize; i++) {
        printf("%d", arr[i]);
        if (i < arraySize - 1) {
            printf(",");
        }
    }
    return;
}

int main() {
    const int a[] = { 
        1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5,
        1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5,
        1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5,
        1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5,
        1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5,
        1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5,
        1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5,
        1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5,
        1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5,
        1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5,
        1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5,
        1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5,
        1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5,
        1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5,
        1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5,
        1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5,
        1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5,
        1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5,
        1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5,
        1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5,
        1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5,
        1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5, 1, 2, 3, 4, 5,
    };
    const int b[] = { 
        6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0,
        6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0,
        6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0,
        6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0,
        6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0,
        6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0,
        6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0,
        6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0,
        6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0,
        6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0,
        6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0,
        6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0,
        6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0,
        6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0,
        6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0,
        6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0,
        6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0,
        6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0,
        6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0,
        6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0,
        6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0,
        6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0, 6, 7, 8, 9, 0,
    };
    const int ARRAY_SIZE = sizeof(a) / sizeof(int);
    
    const int d[3] = { 1, 2, 3 };
    const int e[3] = { 5, 6, 7 };
    int cudaC[ARRAY_SIZE] = { 0 };
    int cpuC[ARRAY_SIZE] = { 0 };

    printf("Array Size: %d\n", ARRAY_SIZE);
    //printf("Array A: ");
    //printArray(a, ARRAY_SIZE);
    //printf("\n");
    //printf("Array B: ");
    //printArray(b, ARRAY_SIZE);
    printf("\n");

    // Add vectors in series (CPU)
    clock_t seriesStart = clock();
    printf("Series Add Start Time: %d\n", seriesStart);
    addVectors(a, b, cpuC, ARRAY_SIZE);
    clock_t seriesEnd = clock();
    printf("Series Add End Time: %d\n", seriesEnd);
    double time_taken = (seriesEnd - seriesStart);
    printf("Execution time for Serial Add (CPU): %f\n", time_taken);

    // Add vectors in parallel.
    clock_t cudaStart = clock();
    printf("Cuda Add Start Time: %d\n", cudaStart);
    hipError_t cudaStatus = mathWithCuda(cudaC, a, b, ARRAY_SIZE, addKernel);;
    clock_t cudaEnd = clock();
    printf("Cuda Add End Time: %d\n", cudaEnd);
    double time_taken_cuda = (cudaEnd - cudaStart);
    printf("Execution time for Cuda Add (GPU): %f\n", time_taken_cuda);

    // multiply vectors in series (CPU)
    clock_t seriesStartMulti = clock();
    printf("Series Multiplication Start Time: %d\n", seriesStartMulti);
    multiplyVectors(a, b, cpuC, ARRAY_SIZE);
    clock_t seriesEndMulti = clock();
    printf("Series Multiplication End Time: %d\n", seriesEndMulti);
    double time_taken_multi = (seriesEndMulti - seriesStartMulti);
    printf("Execution time for Serial Multiplication (CPU): %f\n", time_taken_multi);

    // Multiply vectors in parallel.
    clock_t cudaStartMulti = clock();
    printf("Cuda Multiplication Start Time: %d\n", cudaStartMulti);
    cudaStatus = mathWithCuda(cudaC, a, b, ARRAY_SIZE, multiplyKernel);;
    clock_t cudaEndMulti = clock();
    printf("Cuda Multiplication End Time: %d\n", cudaEndMulti);
    double time_taken_cuda_multi = (cudaEndMulti - cudaStartMulti);
    printf("Execution time for Cuda Multiplication (GPU): %f\n", time_taken_cuda_multi);
    
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    if (cudaTearDown(cudaStatus)) {
        printf("CUDA Teardown Failed!");
        return 1;
    }

    return 0;
}